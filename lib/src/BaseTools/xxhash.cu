#include "hip/hip_runtime.h"
/*
 * xxHash - Extremely Fast Hash algorithm
 * Header File
 * Copyright (C) 2012-2021 Yann Collet
 *
 * BSD 2-Clause License (https://www.opensource.org/licenses/bsd-license.php)
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above
 *      copyright notice, this list of conditions and the following disclaimer
 *      in the documentation and/or other materials provided with the
 *      distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * You can contact the author at:
 *   - xxHash homepage: https://www.xxhash.com
 *   - xxHash source repository: https://github.com/Cyan4973/xxHash
 */

#include <cstdint>

#include "xxhash.H"

namespace xxhash
{

/* #define instead of static const, to be used as initializers */
#define XXH_PRIME32_1  0x9E3779B1U  /*!< 0b10011110001101110111100110110001 */
#define XXH_PRIME32_2  0x85EBCA77U  /*!< 0b10000101111010111100101001110111 */
#define XXH_PRIME32_3  0xC2B2AE3DU  /*!< 0b11000010101100101010111000111101 */
#define XXH_PRIME32_4  0x27D4EB2FU  /*!< 0b00100111110101001110101100101111 */
#define XXH_PRIME32_5  0x165667B1U  /*!< 0b00010110010101100110011110110001 */

#define XXH_rotl32(x,r) (((x) << (r)) | ((x) >> (32 - (r))))

// Byte swap is the same as used for google cityhash
#if defined(__HIPCC__)

__host__ __device__ inline constexpr uint32_t
bswap_32(const uint32_t a_x)
{
  return __byte_perm(a_x, 0, 0x0123);
}

#else

#include <byteswap.h>

#endif
#define XXH_swap32 bswap_32

// Portable and safe solution. Generally efficient.
HOSTDEVICE static xxh_u32
XXH_read32(const void* memPtr)
{
    xxh_u32 val;
    std::memcpy(&val, memPtr, sizeof(val));
    return val;
}

// Normal stripe processing routine.
HOSTDEVICE static xxh_u32
XXH32_round(xxh_u32 acc, xxh_u32 input)
{
    acc += input * XXH_PRIME32_2;
    acc  = XXH_rotl32(acc, 13);
    acc *= XXH_PRIME32_1;
#if (defined(__SSE4_1__) || defined(__aarch64__)) && !defined(XXH_ENABLE_AUTOVECTORIZE)
    XXH_COMPILER_GUARD(acc);
#endif
    return acc;
}

//
HOSTDEVICE XXH_FORCE_INLINE xxh_u32
XXH_readLE32(const void* ptr)
{
    return XXH_CPU_LITTLE_ENDIAN ? XXH_read32(ptr) : XXH_swap32(XXH_read32(ptr));
}

#define XXH_get32bits(p) XXH_readLE32_align(p, align)
//
HOSTDEVICE XXH_FORCE_INLINE xxh_u32
XXH_readLE32_align(const void* ptr, XXH_alignment align)
{
    if (align==XXH_unaligned) {
        return XXH_readLE32(ptr);
    } else {
        return XXH_CPU_LITTLE_ENDIAN ? *(const xxh_u32*)ptr : XXH_swap32(*(const xxh_u32*)ptr);
    }
}

// Mixes all bits to finalize the hash.
HOSTDEVICE static xxh_u32 XXH32_avalanche(xxh_u32 h32)
{
    h32 ^= h32 >> 15;
    h32 *= XXH_PRIME32_2;
    h32 ^= h32 >> 13;
    h32 *= XXH_PRIME32_3;
    h32 ^= h32 >> 16;
    return(h32);
}

// Processes the last 0-15 bytes of @p ptr.
HOSTDEVICE static xxh_u32
XXH32_finalize(xxh_u32 h32, const xxh_u8* ptr, size_t len, XXH_alignment align)
{
#define XXH_PROCESS1 do {                           \
    h32 += (*ptr++) * XXH_PRIME32_5;                \
    h32 = XXH_rotl32(h32, 11) * XXH_PRIME32_1;      \
} while (0)

#define XXH_PROCESS4 do {                           \
    h32 += XXH_get32bits(ptr) * XXH_PRIME32_3;      \
    ptr += 4;                                   \
    h32  = XXH_rotl32(h32, 17) * XXH_PRIME32_4;     \
} while (0)

    if (ptr==NULL) XXH_ASSERT(len == 0);

    /* Compact rerolled version; generally faster */
    if (!XXH32_ENDJMP) {
        len &= 15;
        while (len >= 4) {
            XXH_PROCESS4;
            len -= 4;
        }
        while (len > 0) {
            XXH_PROCESS1;
            --len;
        }
        return XXH32_avalanche(h32);
    } else {
         switch(len&15) /* or switch(bEnd - p) */ {
           case 12:      XXH_PROCESS4;
                         XXH_FALLTHROUGH;
           case 8:       XXH_PROCESS4;
                         XXH_FALLTHROUGH;
           case 4:       XXH_PROCESS4;
                         return XXH32_avalanche(h32);

           case 13:      XXH_PROCESS4;
                         XXH_FALLTHROUGH;
           case 9:       XXH_PROCESS4;
                         XXH_FALLTHROUGH;
           case 5:       XXH_PROCESS4;
                         XXH_PROCESS1;
                         return XXH32_avalanche(h32);

           case 14:      XXH_PROCESS4;
                         XXH_FALLTHROUGH;
           case 10:      XXH_PROCESS4;
                         XXH_FALLTHROUGH;
           case 6:       XXH_PROCESS4;
                         XXH_PROCESS1;
                         XXH_PROCESS1;
                         return XXH32_avalanche(h32);

           case 15:      XXH_PROCESS4;
                         XXH_FALLTHROUGH;
           case 11:      XXH_PROCESS4;
                         XXH_FALLTHROUGH;
           case 7:       XXH_PROCESS4;
                         XXH_FALLTHROUGH;
           case 3:       XXH_PROCESS1;
                         XXH_FALLTHROUGH;
           case 2:       XXH_PROCESS1;
                         XXH_FALLTHROUGH;
           case 1:       XXH_PROCESS1;
                         XXH_FALLTHROUGH;
           case 0:       return XXH32_avalanche(h32);
        }
        XXH_ASSERT(0);
        return h32;   /* reaching this point is deemed impossible */
    }
}

// The implementation for @ref XXH32().
HOSTDEVICE XXH_FORCE_INLINE xxh_u32
XXH32_endian_align(const xxh_u8* input, size_t len, xxh_u32 seed, XXH_alignment align)
{
    xxh_u32 h32;

    if (input==NULL) XXH_ASSERT(len == 0);

    if (len>=16) {
        const xxh_u8* const bEnd = input + len;
        const xxh_u8* const limit = bEnd - 15;
        xxh_u32 v1 = seed + XXH_PRIME32_1 + XXH_PRIME32_2;
        xxh_u32 v2 = seed + XXH_PRIME32_2;
        xxh_u32 v3 = seed + 0;
        xxh_u32 v4 = seed - XXH_PRIME32_1;

        do {
            v1 = XXH32_round(v1, XXH_get32bits(input)); input += 4;
            v2 = XXH32_round(v2, XXH_get32bits(input)); input += 4;
            v3 = XXH32_round(v3, XXH_get32bits(input)); input += 4;
            v4 = XXH32_round(v4, XXH_get32bits(input)); input += 4;
        } while (input < limit);

        h32 = XXH_rotl32(v1, 1)  + XXH_rotl32(v2, 7)
            + XXH_rotl32(v3, 12) + XXH_rotl32(v4, 18);
    } else {
        h32  = seed + XXH_PRIME32_5;
    }

    h32 += (xxh_u32)len;

    return XXH32_finalize(h32, input, len&15, align);
}

/*!
 * @brief Calculates the 32-bit hash of @p input using xxHash32.
 *
 * Speed on Core 2 Duo @ 3 GHz (single thread, SMHasher benchmark): 5.4 GB/s
 *
 * @param input The block of data to be hashed, at least @p length bytes in size.
 * @param length The length of @p input, in bytes.
 * @param seed The 32-bit seed to alter the hash's output predictably.
 *
 * @pre
 *   The memory between @p input and @p input + @p length must be valid,
 *   readable, contiguous memory. However, if @p length is `0`, @p input may be
 *   `NULL`. In C++, this also must be *TriviallyCopyable*.
 *
 * @return The calculated 32-bit hash value.
 *
 * @see
 *    XXH64(), XXH3_64bits_withSeed(), XXH3_128bits_withSeed(), XXH128():
 *    Direct equivalents for the other variants of xxHash.
 * @see
 *    XXH32_createState(), XXH32_update(), XXH32_digest(): Streaming version.
 */
HOSTDEVICE XXH_PUBLIC_API XXH32_hash_t
XXH32 (const void* input, size_t len, XXH32_hash_t seed)
{
    if (XXH_FORCE_ALIGN_CHECK) {
        if ((((size_t)input) & 3) == 0) {   /* Input is 4-bytes aligned, leverage the speed benefit */
            return XXH32_endian_align((const xxh_u8*)input, len, seed, XXH_aligned);
    }   }

    return XXH32_endian_align((const xxh_u8*)input, len, seed, XXH_unaligned);
}

}  // namespace xxhash
